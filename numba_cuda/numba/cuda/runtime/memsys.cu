#include "hip/hip_runtime.h"
#include "memsys.cuh"

extern "C" __global__ void NRT_MemSys_set(NRT_MemSys *memsys_ptr)
{
  TheMSys = memsys_ptr;
}

extern "C" __global__ void NRT_MemSys_read(uint64_t *managed_memsys)
{
  managed_memsys[0] = TheMSys->stats.alloc;
  managed_memsys[1] = TheMSys->stats.free;
  managed_memsys[2] = TheMSys->stats.mi_alloc;
  managed_memsys[3] = TheMSys->stats.mi_free;
}

extern "C" __global__ void NRT_MemSys_init(void)
{
  TheMSys->stats.enabled = false;
  TheMSys->stats.alloc = 0;
  TheMSys->stats.free = 0;
  TheMSys->stats.mi_alloc = 0;
  TheMSys->stats.mi_free = 0;
}

extern "C" __global__ void NRT_MemSys_enable(void)
{
  TheMSys->stats.enabled = true;
}

extern "C" __global__ void NRT_MemSys_disable(void)
{
  TheMSys->stats.enabled = false;
}

extern "C" __global__ void NRT_MemSys_print(void)
{
  if (TheMSys != nullptr)
  {
    printf("TheMSys->stats.enabled %d\n", TheMSys->stats.enabled);
    printf("TheMSys->stats.alloc %d\n", TheMSys->stats.alloc);
    printf("TheMSys->stats.free %d\n", TheMSys->stats.free);
    printf("TheMSys->stats.mi_alloc %d\n", TheMSys->stats.mi_alloc);
    printf("TheMSys->stats.mi_free %d\n", TheMSys->stats.mi_free);
  } else {
    printf("TheMsys is null.\n");
  }
}