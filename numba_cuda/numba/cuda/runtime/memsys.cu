#include "hip/hip_runtime.h"
#include "memsys.cuh"

__device__ size_t memsys_size = sizeof(NRT_MemSys);

namespace detail
{
  void __device__ check_memsys()
  {
    if (TheMSys == nullptr)
    {
      assert(false && "TheMSys pointer is null. Please use NRT_MemSys_set to set pointer first.");
    }
  }
}

extern "C" __global__ void NRT_MemSys_set(NRT_MemSys *memsys_ptr)
{
  TheMSys = memsys_ptr;
}

extern "C" __global__ void NRT_MemSys_read(uint64_t *managed_memsys)
{
  detail::check_memsys();
  managed_memsys[0] = TheMSys->stats.alloc;
  managed_memsys[1] = TheMSys->stats.free;
  managed_memsys[2] = TheMSys->stats.mi_alloc;
  managed_memsys[3] = TheMSys->stats.mi_free;
}

extern "C" __global__ void NRT_MemSys_read_alloc(uint64_t *managed_result)
{
  detail::check_memsys();
  managed_result[0] = TheMSys->stats.alloc;
}

extern "C" __global__ void NRT_MemSys_read_free(uint64_t *managed_result)
{
  detail::check_memsys();
  managed_result[0] = TheMSys->stats.free;
}

extern "C" __global__ void NRT_MemSys_read_mi_alloc(uint64_t *managed_result)
{
  detail::check_memsys();
  managed_result[0] = TheMSys->stats.mi_alloc;
}

extern "C" __global__ void NRT_MemSys_read_mi_free(uint64_t *managed_result)
{
  detail::check_memsys();
  managed_result[0] = TheMSys->stats.mi_free;
}

extern "C" __global__ void NRT_MemSys_init(void)
{
  detail::check_memsys();
  TheMSys->stats.enabled = false;
  TheMSys->stats.alloc = 0;
  TheMSys->stats.free = 0;
  TheMSys->stats.mi_alloc = 0;
  TheMSys->stats.mi_free = 0;
}

extern "C" __global__ void NRT_MemSys_enable_stats(void)
{
  detail::check_memsys();
  TheMSys->stats.enabled = true;
}

extern "C" __global__ void NRT_MemSys_disable_stats(void)
{
  detail::check_memsys();
  TheMSys->stats.enabled = false;
}

extern "C" __global__ void NRT_MemSys_stats_enabled(uint8_t *enabled)
{
  detail::check_memsys();
  *enabled = static_cast<uint8_t>(TheMSys->stats.enabled);
}

extern "C" __global__ void NRT_MemSys_print(void)
{
  if (TheMSys != nullptr)
  {
    printf("TheMSys->stats.enabled %d\n", TheMSys->stats.enabled);
    printf("TheMSys->stats.alloc %lu\n", TheMSys->stats.alloc.load());
    printf("TheMSys->stats.free %lu\n", TheMSys->stats.free.load());
    printf("TheMSys->stats.mi_alloc %lu\n", TheMSys->stats.mi_alloc.load());
    printf("TheMSys->stats.mi_free %lu\n", TheMSys->stats.mi_free.load());
  } else {
    printf("TheMsys is null.\n");
  }
}