/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-2-Clause
 */

// Not all CUDA includes are safe to include in device code compiled by NVRTC,
// because it does not have paths to all system include directories. Headers
// such as cuda_device_runtime_api.h are safe to use in NVRTC without adding
// additional includes.

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
